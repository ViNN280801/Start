#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "GpuProcessing.hpp"

__global__ void processParticlesKernel(Particle *d_particles, size_t num_particles, CubicGrid *d_cubicGrid,
                                       double *d_nodeChargeDensityMap, double timeStep)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_particles)
        return;

    Particle &particle = d_particles[idx];

    // Example of processing - update particle position or charge.
    // This is a simplified version; you can add more logic as per your needs.
    auto meshParams = d_cubicGrid->getTetrahedronsByGridIndex(d_cubicGrid->getGridIndexByPosition(particle.getCentre()));

    for (auto const &meshParam : meshParams)
    {
        if (Mesh::isPointInsideTetrahedron(particle.getCentre(), meshParam.tetrahedron))
        {
            // Update particle charge density or position
            // Use atomic operations to safely update shared variables
            atomicAdd(&d_nodeChargeDensityMap[meshParam.globalTetraId], particle.getCharge());
        }
    }

    // Update particle position
    particle.updatePosition(timeStep);
}

void processParticleTrackerOnGPU(std::vector<Particle> &particles, size_t start_index, size_t end_index,
                                 std::shared_ptr<CubicGrid> cubicGrid,
                                 std::shared_ptr<GSMAssemblier> assemblier,
                                 std::map<size_t, double> &nodeChargeDensityMap, double time)
{
    size_t num_particles = end_index - start_index;
    if (num_particles == 0)
        return;

    // Allocate memory on GPU for particles
    Particle *d_particles;
    hipMalloc(&d_particles, num_particles * sizeof(Particle));

    // Copy particles to the GPU
    hipMemcpy(d_particles, particles.data() + start_index, num_particles * sizeof(Particle), hipMemcpyHostToDevice);

    // Copy cubicGrid to the GPU
    CubicGrid *d_cubicGrid;
    hipMalloc(&d_cubicGrid, sizeof(CubicGrid));
    hipMemcpy(d_cubicGrid, cubicGrid.get(), sizeof(CubicGrid), hipMemcpyHostToDevice);

    // Allocate and initialize memory for nodeChargeDensityMap on the GPU
    double *d_nodeChargeDensityMap;
    hipMalloc(&d_nodeChargeDensityMap, nodeChargeDensityMap.size() * sizeof(double));
    hipMemset(d_nodeChargeDensityMap, 0, nodeChargeDensityMap.size() * sizeof(double));

    // Define grid and block dimensions for CUDA
    int blockSize = 256;
    int numBlocks = (num_particles + blockSize - 1) / blockSize;

    // Launch the kernel on GPU
    processParticlesKernel<<<numBlocks, blockSize>>>(d_particles, num_particles, d_cubicGrid, d_nodeChargeDensityMap, time);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy back the results from the GPU
    std::vector<double> temp_nodeChargeDensityMap(nodeChargeDensityMap.size());
    hipMemcpy(temp_nodeChargeDensityMap.data(), d_nodeChargeDensityMap, nodeChargeDensityMap.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Transfer the GPU results to the host nodeChargeDensityMap
    for (size_t i = 0; i < nodeChargeDensityMap.size(); ++i)
    {
        nodeChargeDensityMap[i] = temp_nodeChargeDensityMap[i];
    }

    // Free GPU memory
    hipFree(d_particles);
    hipFree(d_cubicGrid);
    hipFree(d_nodeChargeDensityMap);
}