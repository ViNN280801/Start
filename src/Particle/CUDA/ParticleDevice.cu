#include <cassert>
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "Particle/CUDA/ParticleDevice.cuh"

ParticleDeviceArray::ParticleDeviceArray(ParticleDevice_t *particles, size_t count)
    : d_particles(particles), count(count) {}

ParticleDeviceArray::~ParticleDeviceArray() { reset(); }

ParticleDeviceArray::ParticleDeviceArray(ParticleDeviceArray &&other) noexcept
    : d_particles(other.d_particles), count(other.count)
{
    other.d_particles = nullptr;
    other.count = 0;
}

ParticleDeviceArray &ParticleDeviceArray::operator=(ParticleDeviceArray &&other) noexcept
{
    if (this != &other)
    {
        if (d_particles)
            hipFree(d_particles);

        d_particles = other.d_particles;
        count = other.count;

        other.d_particles = nullptr;
        other.count = 0;
    }
    return *this;
}

void ParticleDeviceArray::reset()
{
    if (d_particles)
    {
        hipFree(d_particles);
        d_particles = nullptr;
    }
    count = 0ul;
}

bool ParticleDeviceArray::empty() const { return count == 0; }
ParticleDevice_t *ParticleDeviceArray::begin() { return d_particles; }
ParticleDevice_t const *ParticleDeviceArray::cbegin() const { return d_particles; }
ParticleDevice_t *ParticleDeviceArray::end() { return d_particles + count; }
ParticleDevice_t const *ParticleDeviceArray::cend() const { return d_particles + count; }

void ParticleDeviceArray::resize(size_t newCount)
{
    if (d_particles)
        hipFree(d_particles);

    count = newCount;
    hipMalloc(&d_particles, count * sizeof(ParticleDevice_t));
}

ParticleDevice_t &ParticleDeviceArray::operator[](size_t index)
{
    assert(index < count && "Index out of bounds");
    return d_particles[index];
}

ParticleDevice_t const &ParticleDeviceArray::operator[](size_t index) const
{
    assert(index < count && "Index out of bounds");
    return d_particles[index];
}

ParticleDevice_t &ParticleDeviceArray::at(size_t index)
{
    if (index >= count)
        throw std::out_of_range("ParticleDeviceArray::at: index out of range");
    return d_particles[index];
}

ParticleDevice_t const &ParticleDeviceArray::at(size_t index) const
{
    if (index >= count)
        throw std::out_of_range("ParticleDeviceArray::at: index out of range");
    return d_particles[index];
}
