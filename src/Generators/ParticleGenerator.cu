
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <cuda_runtime.h>
#include <curand.h>
#include <curand_kernel.h>

#include "Generators/ParticleGenerator.hpp"
#include "Particle/ParticleDevice.cuh"
#include "Particle/ParticleMemoryConverter.cuh"
#include "Particle/ParticleUtils.hpp"
#include "Utilities/DeviceUtils.cuh"

__global__ void generateParticlesKernel(ParticleDevice_t *particles, size_t count, int type,
                                        double minx, double miny, double minz,
                                        double maxx, double maxy, double maxz,
                                        double minvx, double minvy, double minvz,
                                        double maxvx, double maxvy, double maxvz,
                                        unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    curandState_t state;
    curand_init(seed, idx, 0, &state);

    particles[idx].id = idx;
    particles[idx].type = type;

    // Randomized position
    particles[idx].x = minx + (maxx - minx) * curand_uniform_double(&state);
    particles[idx].y = miny + (maxy - miny) * curand_uniform_double(&state);
    particles[idx].z = minz + (maxz - minz) * curand_uniform_double(&state);

    // Randomized velocity
    particles[idx].vx = minvx + (maxvx - minvx) * curand_uniform_double(&state);
    particles[idx].vy = minvy + (maxvy - minvy) * curand_uniform_double(&state);
    particles[idx].vz = minvz + (maxvz - minvz) * curand_uniform_double(&state);

    double mass = ParticleUtils::getMassFromType(static_cast<ParticleType>(type));
    particles[idx].energy = 0.5 * mass * (particles[idx].vx * particles[idx].vx + particles[idx].vy * particles[idx].vy + particles[idx].vz * particles[idx].vz);
}

START_PARTICLE_VECTOR ParticleGenerator::byVelocities(size_t count, ParticleType type,
                                                      double minx, double miny, double minz,
                                                      double maxx, double maxy, double maxz,
                                                      double minvx, double minvy, double minvz,
                                                      double maxvx, double maxvy, double maxvz)
{
    if (count == 0)
        throw std::logic_error("Cannot generate 0 particles");

    ParticleDeviceArray_t deviceParticles;
    deviceParticles.resize(count);

    int threadsPerBlock = 256;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

    generateParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        deviceParticles.begin(), count, static_cast<int>(type), minx, miny, minz,
        maxx, maxy, maxz, minvx, minvy, minvz, maxvx, maxvy, maxvz, 123'456'789ull);

    cuda_utils::check_cuda_err(cudaGetLastError(), "Error during kernel execution: generateParticlesKernel");
    cuda_utils::check_cuda_err(cudaDeviceSynchronize(), "Failed to synchronize device after particle generation");

    return deviceParticles;
}

__global__ void generateFixedParticlesKernel(ParticleDevice_t *particles, size_t count, int type,
                                             double x, double y, double z,
                                             double vx, double vy, double vz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    particles[idx].id = idx;
    particles[idx].type = type;
    particles[idx].x = x;
    particles[idx].y = y;
    particles[idx].z = z;
    particles[idx].vx = vx;
    particles[idx].vy = vy;
    particles[idx].vz = vz;

    double mass = ParticleUtils::getMassFromType(static_cast<ParticleType>(type));
    particles[idx].energy = 0.5 * mass * (vx * vx + vy * vy + vz * vz);
}

START_PARTICLE_VECTOR ParticleGenerator::byVelocities(size_t count, ParticleType type,
                                                      double x, double y, double z,
                                                      double vx, double vy, double vz)
{
    if (count == 0)
        throw std::logic_error("Cannot generate 0 particles");

    ParticleDeviceArray_t deviceParticles;
    deviceParticles.resize(count);

    int threadsPerBlock = 256;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

    generateFixedParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        deviceParticles.begin(), count, static_cast<int>(type), x, y, z, vx, vy, vz);

    cuda_utils::check_cuda_err(cudaGetLastError(), "Error during kernel execution: generateFixedParticlesKernel");
    cuda_utils::check_cuda_err(cudaDeviceSynchronize(), "Failed to synchronize device after particle generation");

    return deviceParticles;
}

__global__ void generateParticlesWithVelocityModuleKernel(ParticleDevice_t *particles, size_t count, int type,
                                                          double x, double y, double z,
                                                          double v, double maxTheta, double maxPhi,
                                                          unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    curandState_t state;
    curand_init(seed, idx, 0, &state);

    double theta = maxTheta * curand_uniform(&state);
    double phi = maxPhi * curand_uniform(&state);

    double vx = v * sin(theta) * cos(phi);
    double vy = v * sin(theta) * sin(phi);
    double vz = v * cos(theta);

    particles[idx].id = idx;
    particles[idx].type = type;
    particles[idx].x = x;
    particles[idx].y = y;
    particles[idx].z = z;
    particles[idx].vx = vx;
    particles[idx].vy = vy;
    particles[idx].vz = vz;

    double mass = ParticleUtils::getMassFromType(static_cast<ParticleType>(type));
    particles[idx].energy = 0.5 * mass * (vx * vx + vy * vy + vz * vz);
}

START_PARTICLE_VECTOR ParticleGenerator::byVelocityModule(size_t count, ParticleType type,
                                                          double x, double y, double z,
                                                          double v, double maxTheta, double maxPhi)
{
    if (count == 0)
        throw std::logic_error("Cannot generate 0 particles");

    ParticleDeviceArray_t deviceParticles;
    deviceParticles.resize(count);

    int threadsPerBlock = 256;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

    generateParticlesWithVelocityModuleKernel<<<blocksPerGrid, threadsPerBlock>>>(
        deviceParticles.begin(), count, static_cast<int>(type), x, y, z, v, maxTheta, maxPhi, 123'456'789ull);

    cuda_utils::check_cuda_err(cudaGetLastError(), "Error during kernel execution: generateParticlesWithVelocityModuleKernel");
    cuda_utils::check_cuda_err(cudaDeviceSynchronize(), "Failed to synchronize device after particle generation");

    return deviceParticles;
}

__global__ void generateParticlesFromPointSourceKernel(ParticleDevice_t *particles, size_t count,
                                                       double3 position, double energy, int type,
                                                       double expansionAngle, double phi, double theta,
                                                       unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    curandState_t state;
    curand_init(seed, idx, 0, &state);

    // Initialize particle attributes
    particles[idx].id = idx;
    particles[idx].type = type;
    particles[idx].x = position.x;
    particles[idx].y = position.y;
    particles[idx].z = position.z;

    // Set particle energy and direction based on angles
    double vx = energy * sin(theta) * cos(phi);
    double vy = energy * sin(theta) * sin(phi);
    double vz = energy * cos(theta);

    particles[idx].vx = vx;
    particles[idx].vy = vy;
    particles[idx].vz = vz;
    particles[idx].energy = energy;
}

START_PARTICLE_VECTOR ParticleGenerator::fromPointSource(const std::vector<point_source_t> &source)
{
    if (source.empty())
        throw std::logic_error("Point source list is empty");

    size_t totalParticles = 0;
    for (const auto &sourceData : source)
    {
        if (sourceData.count == 0)
            throw std::logic_error("Cannot generate 0 particles from a point source");
        totalParticles += sourceData.count;
    }

    ParticleDeviceArray_t deviceParticles;
    deviceParticles.resize(totalParticles);

    size_t offset = 0;
    for (const auto &sourceData : source)
    {
        double3 position = {sourceData.baseCoordinates[0], sourceData.baseCoordinates[1], sourceData.baseCoordinates[2]};
        int type = static_cast<int>(util::getParticleTypeFromStrRepresentation(sourceData.type));
        size_t count = sourceData.count;
        double energy = sourceData.energy;
        unsigned long long seed = 123'456'789ull;

        int threadsPerBlock = 256;
        int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

        generateParticlesFromPointSourceKernel<<<blocksPerGrid, threadsPerBlock>>>(
            deviceParticles.begin() + offset, count, position, energy, type,
            sourceData.expansionAngle, sourceData.phi, sourceData.theta, seed);

        cuda_utils::check_cuda_err(cudaGetLastError(), "Error during point source particle generation");
        offset += count;
    }

    cuda_utils::check_cuda_err(cudaDeviceSynchronize(), "Failed to synchronize after point source generation");

    return deviceParticles;
}

__global__ void generateParticlesFromSurfaceSourceKernel(ParticleDevice_t *particles, size_t count,
                                                         double3 *cellCenters, double3 *normals,
                                                         double energy, int type, size_t numCells,
                                                         unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;

    size_t cellIdx = idx % numCells;
    particles[idx].id = idx;
    particles[idx].type = type;
    particles[idx].x = cellCenters[cellIdx].x;
    particles[idx].y = cellCenters[cellIdx].y;
    particles[idx].z = cellCenters[cellIdx].z;

    // Calculate angles from normals for direction
    double theta = acos(normals[cellIdx].z / sqrt(normals[cellIdx].x * normals[cellIdx].x +
                                                  normals[cellIdx].y * normals[cellIdx].y +
                                                  normals[cellIdx].z * normals[cellIdx].z));
    double phi = atan2(normals[cellIdx].y, normals[cellIdx].x);

    double vx = energy * sin(theta) * cos(phi);
    double vy = energy * sin(theta) * sin(phi);
    double vz = energy * cos(theta);

    particles[idx].vx = vx;
    particles[idx].vy = vy;
    particles[idx].vz = vz;
    particles[idx].energy = energy;
}

START_PARTICLE_VECTOR ParticleGenerator::fromSurfaceSource(const std::vector<surface_source_t> &source)
{
    if (source.empty())
        throw std::logic_error("Surface source list is empty");

    size_t totalParticles = 0;
    size_t totalCells = 0;
    for (const auto &sourceData : source)
    {
        if (sourceData.count == 0)
            throw std::logic_error("Cannot generate 0 particles from a surface source");
        totalParticles += sourceData.count;
        totalCells += sourceData.baseCoordinates.size();
    }

    ParticleDeviceArray_t deviceParticles;
    deviceParticles.resize(totalParticles);

    std::vector<double3> cellCenters, normals;
    for (const auto &sourceData : source)
    {
        for (const auto &item : sourceData.baseCoordinates)
        {
            std::istringstream iss(item.first);
            double x, y, z;
            iss >> x >> y >> z;
            double3 center = make_double3(x, y, z);

            cellCenters.push_back(center);
            normals.push_back(make_double3(item.second[0], item.second[1], item.second[2]));
        }
    }

    double3 *d_cellCenters = nullptr;
    double3 *d_normals = nullptr;
    cuda_utils::check_cuda_err(cudaMalloc(&d_cellCenters, cellCenters.size() * sizeof(double3)), "Failed to allocate cell centers");
    cuda_utils::check_cuda_err(cudaMalloc(&d_normals, normals.size() * sizeof(double3)), "Failed to allocate normals");

    cudaMemcpy(d_cellCenters, cellCenters.data(), cellCenters.size() * sizeof(double3), cudaMemcpyHostToDevice);
    cudaMemcpy(d_normals, normals.data(), normals.size() * sizeof(double3), cudaMemcpyHostToDevice);

    size_t particleOffset = 0;
    for (const auto &sourceData : source)
    {
        size_t count = sourceData.count;
        size_t numCells = sourceData.baseCoordinates.size();
        double energy = sourceData.energy;
        unsigned long long seed = 123'456'789ull;

        int threadsPerBlock = 256;
        int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

        generateParticlesFromSurfaceSourceKernel<<<blocksPerGrid, threadsPerBlock>>>(
            deviceParticles.begin() + particleOffset, count, d_cellCenters, d_normals,
            energy, static_cast<int>(util::getParticleTypeFromStrRepresentation(sourceData.type)), numCells, seed);

        cuda_utils::check_cuda_err(cudaGetLastError(), "Error during surface source particle generation");
        particleOffset += count;
    }

    cuda_utils::check_cuda_err(cudaDeviceSynchronize(), "Failed to synchronize after surface source generation");

    cudaFree(d_cellCenters);
    cudaFree(d_normals);

    return deviceParticles;
}

#endif // !USE_CUDA
